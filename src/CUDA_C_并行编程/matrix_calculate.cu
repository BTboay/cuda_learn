
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define DIM 10
__global__ void kernel(int *matrix){
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    matrix[offset] = x * y;
}

int main(void){
    int *dev_matrix;
    int matrix[DIM*DIM];
    hipMalloc((void**)&dev_matrix, DIM*DIM*sizeof(int));
    //dim3 grid(DIM, DIM)
    dim3 grid(DIM, DIM, 1);
    kernel<<<grid, 1>>>(dev_matrix);
    hipMemcpy(matrix, dev_matrix, DIM*DIM*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_matrix);
    for (int i = 0; i < DIM; ++i){
        for (int j = 0; j < DIM; ++j){
            printf("%d ", matrix[i + j * DIM]);
        }
        printf("\n");
    }
}
